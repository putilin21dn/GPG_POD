#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void subtraction(double* a, double* b, double* result, long long n) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;
    while(idx < n) {
        result[idx] = a[idx] - b[idx];
        idx += offset;
    }

}


int main(){

  long long n,i;
  scanf("%llu", &n);

  double* a = (double*)malloc(n * sizeof(double));
  double* b = (double*)malloc(n * sizeof(double));
  double* result = (double*)malloc(n * sizeof(double));

  for(i=0; i<n; ++i){
    scanf("%lf",&a[i]);
  }

  for(i=0; i<n; ++i){
    scanf("%lf",&b[i]);
  }

  double *gpu_a, *gpu_b, *gpu_result;
  hipMalloc(&gpu_a, n * sizeof(double));
  hipMalloc(&gpu_b, n * sizeof(double));
  hipMalloc(&gpu_result, n * sizeof(double));

  hipMemcpy(gpu_a, a, n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(gpu_b, b, n * sizeof(double), hipMemcpyHostToDevice);


  subtraction<<<1024, 1024>>>(gpu_a, gpu_b, gpu_result, n);

  hipMemcpy(result, gpu_result, n * sizeof(double), hipMemcpyDeviceToHost);

  for(i=0; i<n; ++i){
    printf("%.10lf ", result[i]);
  }

  free(a);
  free(b);
  free(result);

  hipFree(gpu_a);
  hipFree(gpu_b);
  hipFree(gpu_result);

  return 0;
}